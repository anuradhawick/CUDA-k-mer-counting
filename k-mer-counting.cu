#include "hip/hip_runtime.h"
/*
Copyright 2022 Anuradha Wickramarachchi (anuradhawick@gmail.com)

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

*/

#include <zlib.h>
#include <iostream>
#include <omp.h>
#include <atomic>
#include <vector>
#include <algorithm>

#include "kseq.h"

using namespace std;

KSEQ_INIT(gzFile, gzread)
const uint64_t K_SIZE = 15;
const uint64_t K_MER_COUNT = (uint64_t)pow(4, K_SIZE);
const uint64_t K_MER_MASK = (uint64_t)pow(4, K_SIZE) - 1;
// const char *GENOME = "Homo_sapiens.GRCh38.dna_sm.toplevel.fa.gz";
const char *GENOME = "GRCH38.fasta";
// const char *GENOME = "/media/anuvini/98C4876BC4874B08/lrb_extension/set_100/reads.fasta";


__global__ void CUDA_count_k_mers(uint32_t *k_mer_counts, char *seq, uint64_t len)
{
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len - K_SIZE)
    {
        uint64_t val = 0;
        for (size_t j = 0; j < K_SIZE; j++)
        {
            char s = seq[i + j];
            if (s >= 97 && s <= 122)
            {
                s = s - 32;
            }
            const uint64_t bits = ((s >> 1) & 3);
            if (s == 'A' || s == 'C' || s == 'G' || s == 'T')
            {
                val <<= 2;
                val += bits;
            }
            else
            {
                break;
            }

            if (j == K_SIZE - 1)
            {
                atomicAdd(&k_mer_counts[val], 1);
            }
        }
    }
}

void count_k_mers(vector<atomic<uint32_t>> &k_mer_counts, const char *seq, const size_t &len)
{
#pragma omp parallel for num_threads(32)
    for (size_t i = 0; i < len - K_SIZE; i++)
    {
        uint64_t val = 0;
        for (size_t j = 0; j < K_SIZE; j++)
        {
            const char s = toupper(seq[i + j]);
            const uint64_t bits = ((s >> 1) & 3);
            if (s == 'A' || s == 'C' || s == 'G' || s == 'T')
            {
                val <<= 2;
                val += bits;
            }
            else
            {
                break;
            }

            if (j == K_SIZE - 1)
            {
                uint32_t oval = k_mer_counts[val];
                while (!k_mer_counts[val].compare_exchange_weak(oval, oval + (uint32_t)1))
                {
                };
            }
        }
    }
}

// nvcc k-mer-counting.cu -o kmercounting -lz --dopt=on -gencode arch=compute_80,code=sm_80 -Xcompiler -fopenmp -O3
// intro https://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
// possible optimizations https://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
int main(int argc, char **argv)
{
    gzFile fp;
    kseq_t *seq;
    uint64_t count = 0;
    vector<uint32_t> cpu_k_mer_counts;
    uint32_t *gpu_k_mer_counts;

    if (argc == 2 && string(argv[1]) == "assert")
    {
        cout << "Running the test case" << endl;
    }

    if (argc == 1 || string(argv[1]) == "cpu" || string(argv[1]) == "assert")
    {
        cout << "Running CPU mode" << endl;
        vector<atomic<uint32_t>> k_mer_counts(K_MER_COUNT);

        fp = gzopen(GENOME, "r");
        seq = kseq_init(fp);

        while (kseq_read(seq) >= 0)
        {
            count_k_mers(k_mer_counts, seq->seq.s, seq->seq.l);
            cout << "Counted : " << ++count << "                 \r" << flush;
        }
        cout << "Counted : " << count << endl;

        if ((argc == 2 && string(argv[1]) == "assert"))
        {
            transform(k_mer_counts.begin(), k_mer_counts.end(), back_inserter(cpu_k_mer_counts), [](atomic<uint32_t> &i)
                      { return i.load(memory_order_relaxed); });
        }

        kseq_destroy(seq);
        gzclose(fp);
    }
    if (argc == 2 && string(argv[1]) == "cuda" || argc == 2 && string(argv[1]) == "assert")
    {
        count = 0;
        cout << "Running CUDA mode" << endl;
        uint32_t *k_mer_counts, *cuda_k_mer_counts;
        k_mer_counts = (uint32_t *)malloc(K_MER_COUNT * sizeof(uint32_t));
        for (size_t i = 0; i < K_MER_COUNT; i++)
        {
            k_mer_counts[i] = 0;
        }
        hipMalloc((void **)&cuda_k_mer_counts, K_MER_COUNT * sizeof(uint32_t));
        hipMemcpy(cuda_k_mer_counts, k_mer_counts, K_MER_COUNT * sizeof(uint32_t), hipMemcpyHostToDevice);

        fp = gzopen(GENOME, "r");
        seq = kseq_init(fp);
        char *cuda_seq;
        hipMalloc((void **)&cuda_seq, 1024 * 1024 * 1024 * sizeof(char));

        while (kseq_read(seq) >= 0)
        {
            hipMemcpy(cuda_seq, seq->seq.s, seq->seq.l * sizeof(char), hipMemcpyHostToDevice);
            CUDA_count_k_mers<<<1024 * 1024, 1024>>>(cuda_k_mer_counts, cuda_seq, seq->seq.l);
            cout << "Counted : " << ++count << "                 \r" << flush;
        }
        hipFree(cuda_seq);
        hipMemcpy(k_mer_counts, cuda_k_mer_counts, K_MER_COUNT * sizeof(uint32_t), hipMemcpyDeviceToHost);
        cout << "Counted : " << count << endl;

        kseq_destroy(seq);
        gzclose(fp);
        if ((argc == 2 && string(argv[1]) == "assert"))
        {
            gpu_k_mer_counts = k_mer_counts;
        }
        else
        {
            free(k_mer_counts);
        }
    }
    if (argc == 2 && string(argv[1]) == "assert")
    {
        for (size_t i = 0; i < K_MER_COUNT; i++)
        {
            if (gpu_k_mer_counts[i] != cpu_k_mer_counts[i])
            {
                cout << i << " " << gpu_k_mer_counts[i] << " " << cpu_k_mer_counts[i] << endl;
                cout << "Operation Failed" << endl;
                return -1;
            }
        }
        cout << "Test passed" << endl;
        free(gpu_k_mer_counts);
    }
    return 0;
}